#include "hip/hip_runtime.h"
#include "sys/time.h"
#include "time.h"
#include "global_fun.h"
#include <limits.h>
#include <stdint.h>
// Threads par bloc
#define THREADS_PER_BLOCK 1024
// 0 pour addition, 1 pour multiplication
#define OPERATION 1
#define NON_CONTIGUE_ALIGNE 0
#define TEST 0
#define TMP 0
#define DIVIDE 1000
/* Nombre de CUDA Cores sur GPU:
GPU1: nVidia GeForce GTX TITAN 2688 Cuda cores
GPU2: nVidia Tesla K40c 2880 Cuda cores
GPU3: nVidia Tesla P100-PCIe 7168 Cuda cores */
// Adapter ici le nombre de coeurs
#define CUDA_CORES 2880

// Nombre de blocs necessaires pour l'appel de fonction CUDA, le nombre de blocs dépend du degré
//#define NB_BLOCK ((SIZE+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)

// Caractéristique du corps fini auquel appartiennent les coefficients
//#define MOD 65521

// Indique si l'addition doit se faire modulo MOD
#define IFMOD 1

// Nombre d'opérations par thread


// uint64_t bug à partir de 1<<28 environ
// unsigned int bug à partir de 1<<29 environ
sfixn MOD=1009;
sfixn DEG=500000;
sfixn SIZE=DEG+1;
sfixn OPETHD=1;
sfixn NB_BLOCK=(SIZE+THREADS_PER_BLOCK*OPETHD-1)/(THREADS_PER_BLOCK*OPETHD);

// Fonction d'affichage de polynome
void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<=2*(SIZE-1);i++){
		printf("P[%d]=%d \n",i,res[i]);
	}
	printf("\n");
}

// main
int main(){
	sfixn *a, *b,*res,*tmp; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res,*g_tmp; /*Copie des variables sur GPU, g_p*/
	sfixn i, cut,offset;
  sfixn* test_arit;
	clock_t temps_total;
	clock_t temps_malloc_debut,temps_malloc_fin;
	clock_t temps_memcpy_host_to_device_debut,temps_memcpy_host_to_device_fin;
	clock_t temps_init_debut,temps_init_fin;
	clock_t temps_calcul_debut,temps_calcul_fin;
	//clock_t temps_memcpy_device_to_host_debut, temps_memcpy_device_to_host_fin;
	double temps_pourcent_malloc;
	double temps_pourcent_init;
	double temps_pourcent_calcul;
	double temps_pourcent_memcpy_htod;
	//double temps_pourcent_memcpy_dtoh;
	double size=SIZE*sizeof(sfixn);
	/*On alloue les vecteur de coefficients sur le GPU*/	
	temps_malloc_debut=clock();
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	
	/*
	hipDeviceProp_t  prop;
	hipGetDeviceProperties( &prop, 0 );
  int blocks = prop.multiProcessorCount;
  int mem = prop.sharedMemPerBlock;
  printf("prop.multiProcessorCount=%d\n",blocks); 
  int deviceCount;
	hipGetDeviceCount(& deviceCount);
	printf("deviceCount=%d  && mem_partage=%d\n",deviceCount,mem);
	for(i=0;i<3;i++)
		printf("max grid=%d\n",prop.maxGridSize[i]); 
	*/ 
	
	if(OPERATION){
    res = (sfixn*)malloc(2*size);
    memset(res, 0, 2*SIZE*sizeof(sfixn));
    hipMalloc((void**)&g_res, 2*size);
    hipMalloc((void**)&g_tmp,DIVIDE*size);
    tmp = (sfixn*)malloc(DIVIDE*size);
  } else { 
    res = (sfixn*)malloc(size);
    hipMalloc((void**)&g_res, size);
  }
  temps_malloc_fin=clock();
	printf("L'allocation mémoire des differents tableaux prend %fs\n",(double)(temps_malloc_fin-temps_malloc_debut)/CLOCKS_PER_SEC);
  offset=0;
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre SIZE */
	temps_init_debut=clock();
	for(i=0; i<SIZE; i++){
		a[i]=(sfixn) ((MOD)*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) ((MOD)*((double)rand())/ RAND_MAX);
		/*a[i]=i%11;
		b[i]=(i+1)%11;*/
		//printf("a[%d]=%d && b[%d]=%d \n",i,a[i],i,b[i]);
	}	
	temps_init_fin=clock();
	printf("L'initialisation des polynomes a et b %fs\n",(double)(temps_init_fin-temps_init_debut)/CLOCKS_PER_SEC);
  if(OPERATION){
	  if(TEST){
	  	test_arit = (sfixn*)malloc(2*size);
	  	memset(test_arit, 0, 2*SIZE*sizeof(sfixn));
	  	test_arit=multiplication_polynome_mod(a,b,MOD,SIZE);
	  	printf("test_arit[%d]=%d\n",0,test_arit[0]); 
	  }
  } else {	
  	if(IFMOD){
  		printf("res[%d]=%d\n",0,(a[0]+b[0])%MOD);    	
  		printf("res[%d]=%d\n",(SIZE-1),(a[SIZE-1]+b[SIZE-1])%MOD);
		}else{
			printf("res[%d]=%d\n",0,(a[0]+b[0]));
			printf("res[%d]=%d\n",(SIZE-1),(a[SIZE-1]+b[SIZE-1]));
		}
  }

	/*Le modulo p un nombre premier*/	

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	temps_memcpy_host_to_device_debut=clock();
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
  if(OPERATION) hipMemcpy(g_res, res, 2*size, hipMemcpyHostToDevice);
	temps_memcpy_host_to_device_fin=clock();
	printf("La copie des tableaux du CPU vers le GPU prend %fs\n",(double)(temps_memcpy_host_to_device_fin-temps_memcpy_host_to_device_debut)/CLOCKS_PER_SEC);
	/*Appel de fonction sur le GPU */
	/* 
	  CUDA_CORES+THREAD_PER_BLOCK-1 : -permet de ne pas rajouter un block si CUDA_CORES est un multiple de THREAD_PER_BLOCK
					  -permet de rajouter un block si CUDA_CORES n'est pas un multiple de THREAD_PER_BLOCK	

		VERSION SIZE op/thread (ERRONE) : 
		mult_mod_multhd<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE);
		
		VERSION 1 op/thread  :
		for(i=0;i<SIZE;i++){
    	mult_mod<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,i);
    }
		
		VERSION OPTHD op/thread  : 
		for(i=0;i<SIZE;i++){
    	mult_mod_multhd2<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,i,OPETHD);
    }
		
		VERSION TMP :
		sfixn cpt,j;
		sfixn nb_iter=SIZE/DIVIDE;
		sfixn nb_block_mul=(DIVIDE+THREADS_PER_BLOCK*OPETHD-1)/(THREADS_PER_BLOCK*OPETHD);
		
		//mult_mod_multhd<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_tmp,SIZE);
		for(cpt=1;cpt<=nb_iter;cpt++){
			//printf("nb_iter = %d\n",nb_iter);
			mult_mod_multhd_tmp<<<nb_block_mul,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_tmp,cpt,DIVIDE,SIZE);
			//hipDeviceSynchronize();
			hipMemcpy(tmp, g_tmp,DIVIDE*size, hipMemcpyDeviceToHost);
			for(i=0;i<SIZE;i++){
				for(j=0;j<DIVIDE;j++){
					//res[i+j+(cpt-1)*DIVIDE]=(res[i+j+(cpt-1)*DIVIDE]+tmp[i+j*SIZE])%MOD;
					res[i+j+(cpt-1)*DIVIDE]=(res[i+j+(cpt-1)*DIVIDE]+tmp[i+j*SIZE]);
					ADD_SCALAR_SHIFT(MOD,res[i+j+(cpt-1)*DIVIDE]);
				}
			}
		}
		
		
		}
	*/
	
	temps_calcul_debut=clock();
  if(OPERATION){
  	printf("NB BLOCK=%d\n",NB_BLOCK);
  	printf("Multiplication naive sur GPU\n");
  		
  	
  	/*sfixn nb_bloc=(2*SIZE-1)/(THREADS_PER_BLOCK);
  	if(((2*SIZE-1)%THREADS_PER_BLOCK)!=0)
  		nb_bloc++;
  	*/
  	
  	/*sfixn nb_bloc=28;
  	printf("nb bloc=%d\n",nb_bloc);
  	sfixn T=(2*SIZE-1)/(THREADS_PER_BLOCK*nb_bloc);
		if((2*SIZE-1)%(THREADS_PER_BLOCK*nb_bloc)!=0)
			T++;
		printf("T=%d\n",T);*/
		
		
		sfixn T=2;
		sfixn nb_bloc=(2*SIZE-1)/(T*THREADS_PER_BLOCK);
  	if(((2*SIZE-1)%THREADS_PER_BLOCK)!=0)
  		nb_bloc++;		
  	//sfixn max_uint=UINT_MAX;
		printf("T=%d et nb_bloc=%d\n",T,nb_bloc);
		sfixn mod2=(MOD-1)*(MOD-1);
		sfixn iter=UINT_MAX/(mod2);
  	printf("UINT_MAX=%u\n",UINT_MAX);
  	printf("iter=%u\n",iter);
  	sfixn max_uint=UINT_MAX-(MOD-1);
  	printf("UINT_MAX-(MOD-1)=%u\n",max_uint);
  	sfixn nb=(2*SIZE)/T;
  	if((2*SIZE)%T!=0)
  		nb++;
  	printf("nb=%d\n",nb);
  	printf("nb_iter=%d\n",iter);
  	//test<<<nb_bloc,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,T);
  	//mult_mod_repart_non_contigue<<<nb_bloc,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,T,nb);
  	//mult_mod_repart_iter<<<nb_bloc,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,T,iter);
  	//test<<<nb_bloc,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,T);
  	//mult_mod_repart<<<nb_bloc,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,T);
  	//mult_mod_multhd<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE);
  	//mult_mod_share<<<NB_BLOCK,THREADS_PER_BLOCK,(2*SIZE-1)*sizeof(sfixn)>>>(g_a,g_b,MOD,g_res,SIZE);
		
		mult_mod_repart<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,NB_BLOCK);
  	
  	/*for(i=0;i<SIZE;i++){
    	mult_mod<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,i);
    }*/
        
    /*for(i=0;i<SIZE;i++){
    	mult_mod_multhd2<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,i,OPETHD);
    }*/
  	
  	//hipDeviceSynchronize();
    //printf("Karatsuba sur GPU\n");
    /*karatsuba<<<1,1>>>(g_a,g_b,MOD,g_res,SIZE);
  
  
  	*/
  	if(NON_CONTIGUE_ALIGNE){
			sfixn e=0;
			sfixn j;
		  sfixn* res1 = (sfixn*)malloc(2*size);
		  memset(res1, 0, 2*size);
		  hipMemcpy(res1, g_res, 2*size-1, hipMemcpyDeviceToHost);
		  for(i=0;i<nb;i++){
		  	for(j=0;j<T;j++){
		  		res[i+j*nb]=res1[e];
		  		e++;
		  	}	
		  }
  	}
  } else {
	  if(NB_BLOCK>CUDA_CORES){
			cut=NB_BLOCK/CUDA_CORES;
			if((NB_BLOCK%CUDA_CORES) != 0){
			  cut++;
			}
			printf("Le nombre de blocs necessaire est de %d.\nIl est superieur au nombre de coeur du GPU qui est de %d.\nOn doit donc diviser les polynomes en %d parties afin de ne pas depasser le nombre de coeurs maximal.\n",NB_BLOCK,CUDA_CORES,cut);
			printf("cut=%d\n",cut);
			if(IFMOD){
			  for(i=0;i<cut;i++){
			     offset=i*CUDA_CORES*THREADS_PER_BLOCK*OPETHD;
			     if(i==cut-1){
						 printf("offset = %d\n",offset);              
						 int nb_block_restant=(((SIZE-offset)+(THREADS_PER_BLOCK*OPETHD)-1)/(THREADS_PER_BLOCK*OPETHD));
						 printf("nb_block_restant = %d\n",nb_block_restant); 
						 add_mod_multhd<<<nb_block_restant,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,offset,OPETHD);
					 }else{
			     	add_mod_multhd <<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,offset,OPETHD);
			 		 }
				}    
		}else{
			printf("IFMOD =0\n");
			for(i=0;i<cut;i++){
			  offset=i*CUDA_CORES*THREADS_PER_BLOCK;
			  if(i==cut-1){
					printf("offset = %d\n",offset);              
					sfixn nb_block_restant2=(((SIZE-offset)+(THREADS_PER_BLOCK*OPETHD)-1)/(THREADS_PER_BLOCK*OPETHD));
					printf("nb_block_restant = %d\n",nb_block_restant2); 
					add<<<nb_block_restant2,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,SIZE,offset);
				}else{
			  	add<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,SIZE,offset);
			  }
			}  
	 	}
	 }else{
  	printf("Le nombre de blocs necessaire est de %d.\nIl est inferieur au nombre de coeur du GPU qui est de %d.\nOn peut donc sommer les coefficients du polynome en 1 seule fois.\n",NB_BLOCK,CUDA_CORES);
	  if(IFMOD==1){
    	add_mod_multhd<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,SIZE,0,OPETHD);
    }else{
      add<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,SIZE,0);
    	}	    
	  }
  }
 	
	//printf("Le calcul prend %fs\n",(double)(temps_calcul_fin-temps_calcul_debut)/CLOCKS_PER_SEC);
	/*Copie du resultat du GPU sur le CPU*/
	//temps_memcpy_device_to_host_debut=clock();
	if(OPERATION && !TMP && !NON_CONTIGUE_ALIGNE){
		hipMemcpy(res, g_res, 2*size-1, hipMemcpyDeviceToHost);
  } 
  if(!OPERATION) {
    hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);
  }
  //temps_memcpy_device_to_host_fin=clock();
	//printf("La copie des tableaux du GPU vers le CPU prend %fs\n",(double)(temps_memcpy_device_to_host_fin-temps_memcpy_device_to_host_debut)/CLOCKS_PER_SEC);
	temps_calcul_fin=clock();
	printf("Le calcul prend %fs\n",(double)(temps_calcul_fin-temps_calcul_debut)/CLOCKS_PER_SEC);
	
	
	/*Liberation de l'espace alloué sur le GPU */
  //affichage_polynome(res);
	if(OPERATION){
		if(TEST){
			printf("P[0]=%d\n",test_arit[0]);
			printf("P[%d]=%d\n",2*(SIZE-1),test_arit[2*(SIZE-1)]);
			printf("TEST\n");
			for(i=0;i<=2*(SIZE-1);i++){
				if(res[i]!=test_arit[i]){
					printf("ERREUR à l'indice i=%d CPU=%d et GPU=%d\n",i,test_arit[i],res[i]);
				}
			}
			free(test_arit);
		}
		//printf("P[0]=%d\n",res[0]);
		/*printf("P[1]=%d\n",res[1]);
	  printf("P[%d]=%d\n",2*(SIZE-1),res[2*(SIZE-1)]);*/
	  	  
	} else {
		printf("P[%d]=%d\n",0,res[0]);
	  printf("P[%d]=%d\n",(SIZE-1),res[(SIZE-1)]);
	  printf("VERIFICATION\n");
	  if(IFMOD){
	  	printf("P[%d]=%d\n",0,(a[0]+b[0])%MOD);
	  	printf("P[%d]=%d\n",(SIZE-1),(a[SIZE-1]+b[SIZE-1])%MOD);
	  }else{
	  	printf("P[%d]=%d\n",0,(a[0]+b[0]));
	  	printf("P[%d]=%d\n",(SIZE-1),(a[SIZE-1]+b[SIZE-1]));
	  }
	}
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	if(TMP){
		hipFree(g_tmp);
		free(tmp);
	}
	free(a);
	free(b);
	free(res);
	
	
	temps_total=clock();
	temps_pourcent_init=(double)(((double)temps_init_fin-(double)temps_init_debut)/(double)temps_total)*100;
	temps_pourcent_malloc=(double)(((double)temps_malloc_fin-(double)temps_malloc_debut)/(double)temps_total)*100;
	temps_pourcent_memcpy_htod=(double)(((double)temps_memcpy_host_to_device_fin-(double)temps_memcpy_host_to_device_debut)/(double)temps_total)*100;
	temps_pourcent_calcul=(double)(((double)temps_calcul_fin-(double)temps_calcul_debut)/(double)temps_total)*100;
	//temps_pourcent_memcpy_dtoh=(double)(((double)temps_memcpy_device_to_host_fin-(double)temps_memcpy_device_to_host_debut)/(double)temps_total)*100;
	printf("----------------REPARTITION TEMPS: -----------------\n");
	printf("L'allocation prend %f %% du temps total\n",(double)temps_pourcent_malloc);
	printf("L'initialisation prend %f %% du temps total\n",(double)temps_pourcent_init);
	printf("La copie des tableaux du CPU vers le GPU prend %f %% du temps total\n",(double)temps_pourcent_memcpy_htod);
	printf("Le calcul prend %f %% du temps total\n",(double)temps_pourcent_calcul);
	//printf("La copie des tableaux du GPU vers le CPU prend %f %% du temps total\n",(double)temps_pourcent_memcpy_dtoh);		
	printf("Le temps_total d'execution est de : %fs\n",(double)temps_total/CLOCKS_PER_SEC);
	
	
	return 0;
}
