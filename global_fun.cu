#include "hip/hip_runtime.h"
#include "global_fun.h"
#include "types.h"

// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	sfixn r;
	if(ind+offset < deg){
		r=(a[ind+offset]+b[ind+offset]);
		r-= p;
   		r += (r >> 31) & p;
		res[ind+offset] = r;
		/*if(r >= p){
			r-=p;
		}
		res[ind+offset] = r;*/
		//res[ind+offset] = (a[ind+offset]+b[ind+offset])%p;


// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	
	if(ind+offset < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
	}
	
}

// Fonction addition modulo MOD de polynomes sur Device (GPU) avec multiples opérations par thread
__global__ void add_mod_multhd(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset, sfixn op_thread){
	sfixn i;
	sfixn ind=op_thread*threadIdx.x+blockIdx.x*blockDim.x*op_thread;  //blockDim.x correspond au nombre de threads par block 
	for(i=0;i<op_thread;i++){
		if(ind+offset+i < deg){
			res[ind+offset+i]=(a[ind+offset+i]+b[ind+offset+i])%p;
			if(ind+offset+i == deg-1){
				printf("res=%d\n",res[ind+offset+i]);
				}
		}
	}
	
}


// Fonction addition de polynomes sans modulo sur Device (GPU) avec multiples opérations par thread
__global__ void add_multhd(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset, sfixn op_thread){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	
	if(ind+offset < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
	}
	
}

