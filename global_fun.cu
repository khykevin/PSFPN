#include "hip/hip_runtime.h"
#include "global_fun.h"
#include "types.h"


// Fonction d'affichage de polynome
__global__ void affiche_polynome(sfixn *res, sfixn size){
	sfixn i;
	for(i=size-10;i<size;i++){
		printf("P[%d]=%d\n",i,res[i]);
	}
	printf("\n");
}



/*
void karatsuba(sfixn* kar, sfixn* a,sfixn* b, sfixn size, sfixn p){
		sfixn *a0;
		sfixn *a1;
		sfixn *b0;
		sfixn *b1;
		sfixn* h0;
		sfixn* h1;
		sfixn* h2;
		sfixn* h1dec;
		sfixn* h2dec;
		sfixn* h0inv;
		sfixn* h2inv;
		sfixn* sum1;
		sfixn* sum2;
		sfixn* suma;
		sfixn* sumb;
		sfixn m=size/2;
	  hipMalloc((void**)&a0,m*sizeof(sfixn));
	  if(a0==NULL){
		  printf("karatsuba : a0\n");
		  exit(0);
	  }
	  hipMalloc((void**)&a1,m*sizeof(sfixn));
	  if(a1==NULL){
		  printf("karatsuba : a1\n");
		  exit(0);
	  }
	  hipMalloc((void**)&b0,m*sizeof(sfixn));
	  if(b0==NULL){
		  printf("karatsuba : b0\n");
		  exit(0);
	  }	
	  hipMalloc((void**)&b1,m*sizeof(sfixn));
	  if(a0==NULL){
		  printf("karatsuba : b1\n");
		  exit(0);
	  }
	  hipMalloc((void**)&h0, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h0inv, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h2inv, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&suma, (m)*sizeof(sfixn));
	  hipMalloc((void**)&sumb, (m)*sizeof(sfixn));
	  hipMalloc((void**)&sum1, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&sum2, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h1, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h2, (2*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h1dec, (3*m-1)*sizeof(sfixn));
	  hipMalloc((void**)&h2dec, (4*m-1)*sizeof(sfixn));
		karatsuba_gpu<<<size/1024,1024>>>(kar, h0,h0inv, h2inv, suma, sumb, sum1, sum2, h1, h2, h1dec, h2dec, a, b, a0, a1, b0, b1, size, p);
		hipFree(a0);
	  hipFree(a1);
	  hipFree(b0);
	  hipFree(b1);
    hipFree(h0);
	  hipFree(h1);
	  hipFree(h2);
	  hipFree(h1dec);
	  hipFree(h2dec);
	  hipFree(sum1);
	  hipFree(sum2);
	  hipFree(h2inv);
	  hipFree(h0inv);
	  hipFree(suma);
	 	hipFree(sumb);
		
}
__global__ void karatsuba_gpu(sfixn* kar, sfixn* h0, sfixn* h0inv, sfixn* h2inv, sfixn* suma, sfixn* sumb, sfixn* sum1, sfixn* sum2, sfixn* h1, sfixn* h2, sfixn* h1dec, sfixn* h2dec,  sfixn* a, sfixn* b, sfixn* a0, 		sfixn* a1,sfixn* b0, sfixn* b1,sfixn size, sfixn p){
	karatsuba_rec(kar, h0,h0inv, h2inv, suma, sumb, sum1, sum2, h1, h2, h1dec, h2dec, a, b, a0, a1, b0, b1, size, p);
}



__device__ void karatsuba_rec(sfixn* kar, sfixn* h0, sfixn* h0inv, sfixn* h2inv, sfixn* suma, sfixn* sumb, sfixn* sum1, sfixn* sum2, sfixn* h1, sfixn* h2, sfixn* h1dec, sfixn* h2dec,  sfixn* a, sfixn* b, sfixn* a0, 		sfixn* a1,sfixn* b0, sfixn* b1,sfixn size, sfixn p){
	sfixn i,m;
	
	if(size==1){
		kar[0]=(a[0]*b[0])%p;
	}else{
	  m=size/2;
	  for(i=0;i<m;i++){
		  //printf("a[%d]=%d && b[q%d]=%d\n",i,a[i],i,b[i]);
		  a0[i]=a[i];	
		  b0[i]=b[i];
	  }
	  for(i=m;i<size;i++){
		  //printf("a[%d]=%d && b[%d]=%d\n",i,a[i],i,b[i]);
		  a1[i-m]=a[i];		
		  b1[i-m]=b[i];
	  }
	  karatsuba_rec(h0,a0,b0,m,p);
	  karatsuba_rec(h2,a1,b1,m,p);
	  somme_void(suma,a0,a1,m,m,p);
	  somme_void(sumb,b0,b1,m,m,p);
	  karatsuba_rec(h1,suma,sumb,m,p);
	  oppose_void(h0inv,h0,2*m-1,p);
	  oppose_void(h2inv,h2,2*m-1,p);
	  somme_void(sum1,h0inv,h2inv,2*m-1,2*m-1,p);
	  somme_void(h1,h1,sum1,2*m-1,2*m-1,p);
	  decalage_void(h1dec,h1,m,2*m-1);
	  decalage_void(h2dec,h2,2*m,2*m-1);
	  somme_void(sum2,h1dec,h2dec,2*m-1+m,4*m-1,p);
	  somme_void(kar,h0,sum2,2*m-1,2*m-1+2*m,p);
	  }
}
*/


__device__ void decalage_void(sfixn* dec, sfixn *tab, sfixn size, sfixn n){
	sfixn i;	
	for(i=0;i<size;i++){
		dec[i]=0;
	}
	for(i=size;i<size+n;i++){
		dec[i]=tab[i-size];
	}	
}

__device__ void somme_void(sfixn* sum, sfixn *a, sfixn *b, sfixn size_a, sfixn size_b, sfixn p){
	sfixn i;
	if(size_a>size_b){
		for(i=0;i<size_a;i++){
			if(i<size_b)
				sum[i]=(a[i]+b[i])%p;
			else 
				sum[i]=a[i];
		}

	}else{
		for(i=0;i<size_b;i++){
			if(i<size_a)
				sum[i]=(a[i]+b[i])%p;
			else 
				sum[i]=b[i];
		}
	}
}


__device__ void oppose_void(sfixn* op, sfixn* tab, sfixn size, sfixn p){
	sfixn i;
	int x,p1;
	p1=(int)p;
	for(i=0;i<size;i++){
	  x=(int)tab[i];
	  x=-x;
	  x=((x%p1)+p1)%p1;
	  
		op[i]=(unsigned int)x;
	}
}



// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn offset){
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
  sfixn r;
  if(ind+offset < size){
    r = a[ind+offset]+b[ind+offset];
		MOD_PERCENT(p,r);
		res[ind+offset] = r;
  }
}

// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn size, sfixn offset){
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
  //printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
  if(ind+offset < size){
    res[ind+offset]=a[ind+offset]+b[ind+offset];
  }   
}
__global__ void mult_mod_multhd_tmp(sfixn* a, sfixn* b, sfixn p, sfixn* tmp, sfixn cpt, sfixn divide, sfixn size){
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
  sfixn i,r;
  if(ind < divide){
	  for(i=0;i<size;i++){
	    r = (a[ind+(cpt-1)*divide]*b[i])%p;
	    //__syncthreads();
      tmp[i+ind*size]=r;
      //if(i==0 && ind==0) printf("r = %d, tmp[0] = %d\n",r,tmp[0]);
      //res[ind+i] += r;
      //MOD_PERCENT(p,res[ind+i]);
      //res[ind+i] = r;
    }
  }
}

/*
__global__ void mult_mod_repart(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<2*size-1){
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}
			}
		}
	}
}
*/


/*
__global__ void mult_mod_repart(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn max_uint){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<2*size-1){
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						if(res[i]>=max_uint)MOD_PERCENT(p,res[i]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						if(res[i]>=max_uint)MOD_PERCENT(p,res[i]);
					}
				}
				MOD_PERCENT(p,res[i]);
			}
		}
	}
}
*/

__global__ void mult_mod_repart(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<=(2*(size-1))){
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						//MOD_PERCENT(p,r);
						res[i]+=r;
						//MOD_PERCENT(p,res[i]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						//MOD_PERCENT(p,r);
						res[i]+=r;
						//MOD_PERCENT(p,res[i]);
					}
				}
			}
		}
	}
}


__global__ void mult_mod_repart_iter(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn iter){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<=(2*(size-1))){
		sfixn i,j,k,r,nb_iter;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					sfixn decal=i-(size-1);
					nb_iter=(2*(size-1)-i)/iter;
					for(j=0;j<nb_iter;j++){
						for(k=j*iter+decal;k<(j+1)*iter+decal;k++){
							r=a[k]*b[i-k];
							res[i]+=r;
						}
						MOD_PERCENT(p,res[i]);
					}
					for(j=nb_iter*iter+decal;j<size;j++){
						r=a[j]*b[i-j];
						res[i]+=r;
					}
					MOD_PERCENT(p,res[i]);
				}else{
					nb_iter=i/iter;
					for(j=0;j<nb_iter;j++){
						for(k=j*iter;k<(j+1)*iter;k++){
							r=a[k]*b[i-k];
							res[i]+=r;
						}
						MOD_PERCENT(p,res[i]);
					}
					for(j=nb_iter*iter;j<=i;j++){
						r=a[j]*b[i-j];
						res[i]+=r;
					}
					MOD_PERCENT(p,res[i]);
				}
			}
		}
	}
}



__global__ void test(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T){	
	//sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
}



__global__ void mult_mod_repart_non_contigue(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn nb){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<=(2*(size-1)/T)){
		sfixn i,j,r;
		for(i=ind;i<=ind+nb*(T-1);i=i+nb){
			if(i==20000)
				printf("ind=%d\n",ind);
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}
			}
		}
	}
}

__global__ void mult_mod_repart_non_contigue2(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn nb){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	sfixn ind_offset=ind*T;
	if(ind<=(2*(size-1)/T)){
		sfixn i,j,r;
		for(i=ind;i<=ind+nb*(T-1);i=i+nb){
			if(i==20000)
				printf("ind=%d\n",ind);
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[ind_offset]+=r;
						MOD_PERCENT(p,res[ind_offset]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[ind_offset]+=r;
						MOD_PERCENT(p,res[ind_offset]);
					}
				}
			}
			ind_offset++;
		}
		
	}
}




__global__ void mult_mod_repart_depassement(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn max_uint, sfixn iter){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<2*size-1){
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					if(i>=iter){
						for(j=i-(size-1);j<size;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
							if(res[i]>=max_uint)MOD_PERCENT(p,res[i]);
						}
					}else{
						for(j=i-(size-1);j<size;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
						}
					}
				}else{
					if(i>=iter){
						for(j=0;j<=i;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
							if(res[i]>=max_uint)MOD_PERCENT(p,res[i]);
						}
					}else{
						for(j=0;j<=i;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
						}
					}
				}
				MOD_PERCENT(p,res[i]);
			}
		}
	}
}



/*
__global__ void mult_mod_repart(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn T, sfixn max_iter){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<2*size-1){
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					if(i>=max_iter){
						for(j=i-(size-1);j<size;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
							MOD_PERCENT(p,res[i]);
							//if(k%(max_iter)==0 || j==size-1)MOD_PERCENT(p,res[i]);
							//k++;
						}
					}else{
						for(j=i-(size-1);j<size;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
							//MOD_PERCENT(p,res[i]);
							//if(k%(max_iter)==0 || j==size-1)MOD_PERCENT(p,res[i]);
							//k++;
						}
						MOD_PERCENT(p,res[i]);	
					}
					
				}else{
					if(i>=max_iter){
						//k=1;
						for(j=0;j<=i;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
							MOD_PERCENT(p,res[i]);
							//if(k==max_iter){
								//MOD_PERCENT(p,res[i]);
								//k=0;
							//}
							//k++;
						}
					}else{
						for(j=0;j<=i;j++){
							r=a[j]*b[i-j];
							MOD_PERCENT(p,r);
							res[i]+=r;
						}
						MOD_PERCENT(p,res[i]);
					}
					
				}
			}
		}
	}
}
*/
/*
__global__ void mult_mod_repart(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn nb_bloc){	
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	if(ind<2*size-1){
		sfixn T=(2*size-1)/(blockDim.x*nb_bloc);
		if((2*size-1)%blockDim.x!=0)
			T++;
		//printf("T=%d\n",T);
		sfixn i,j,r;
		for(i=ind*T;i<(ind+1)*T;i++){
			if(i<2*size-1){
				if(i>=size){
					for(j=i-(size-1);j<size;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}else{
					for(j=0;j<=i;j++){
						r=a[j]*b[i-j];
						MOD_PERCENT(p,r);
						res[i]+=r;
						MOD_PERCENT(p,res[i]);
					}
				}
			}
		}
	}
}
*/


__global__ void mult_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn i){
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
  sfixn r;
  printf("ind=%d\n",ind);
  if(ind < size){
    r = (a[ind]*b[i]);//%p;
    res[ind+i] += r;
    //MOD_PERCENT(p,res[ind+i]);
    //res[ind+i] = (res[ind+i]+(a[ind]*b[i]))%p;
  }
}


__global__ void mult_mod_share(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size){
  extern __shared__ sfixn selfres[];
  sfixn tid=threadIdx.x;
  //sfixn bid=size/blockIdx.x;
  //sfixn bid=blockIdx.x;
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
  sfixn i,r,j;
  //printf("TTTTTTTEEEEEEEEEEEETTTTTT\n");
  if(ind<size){
		for(i=0;i<size;i++){
			if(ind+i==1) printf("avant %d %d\n",selfres[i+ind]);
		  r = (a[ind]*b[i]);
		  __syncthreads();
		  selfres[ind+i]+=r;
		  if(i+ind==1) printf("apres %d %d\n",selfres[i+ind]);
		  //__syncthreads();
		}
		__syncthreads();
		if(tid==0){
			//__syncthreads();
		  for(j=0;j<2*size-1;j++){
		   	//__syncthreads();
		    res[j]=(res[j]+selfres[j]);
		    //__syncthreads();
		  }
		  //__syncthreads();
		}
	}
}

__global__ void mult_mod_multhd(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size){
    sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
    sfixn i,r;
   
    if(ind < size){
        for(i=0;i<size;i++){
            r = (a[ind]*b[i]);//%p;
            __syncthreads();
            res[ind+i] = (res[ind+i]+r);//%p;
        }
    }
}





/*
sfixn i,r,q;
double ninv = 1 / (double)p;
q  = (sfixn) ((((double) a[ind]) * ((double) b[i])) * ninv);
            r = a[ind] * b[i] - q * p;   
            r += (r >> 31) & p;
            r -= p;
        r += (r >> 31) & p;
            r = (res[ind+i]+r);
            r = (res[ind+i]+r);   
            r-= p;
           r += (r >> 31) & p;
            res[ind+i] = r;   
*/


sfixn* multiplication_polynome_mod(sfixn* a, sfixn* b, sfixn p,sfixn size){
    sfixn*res;
    sfixn i,j,r;
    res =(sfixn*)malloc(2*size*sizeof(sfixn));
    memset(res, 0, 2*size*sizeof(sfixn));
    for(i=0;i<size;i++){
        for(j=0;j<size;j++){
            r=(a[i]*b[j]);
            MOD_PERCENT(p,r); 
            r+=res[i+j];
            MOD_PERCENT(p,r);
            res[i+j]=r;
        }           
    }
    return res;
}


__global__ void mult_mod_multhd2(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn i, sfixn op_thread){
  sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
  ind=ind*op_thread;
  sfixn j,r;
  if(ind<size){
		for(j=0;j<op_thread;j++){
		  if(ind+i+j <= 2*(size-1) && ind+j<size){
		    r = (a[ind+j]*b[i])%p;
		    r += res[ind+i+j];
		    MOD_PERCENT(p,r);
		    res[ind+i+j] = r;
		  }
		}
	}
}
/*__global__ void mult_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn i){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;
	sfixn r;
	if(ind < size){
		r = (a[ind]*b[i])%p;
		res[ind+i] = (res[ind+i]+r)%p;
		//res[ind+i] = (res[ind+i]+(a[ind]*b[i]))%p;
	}
}*/


// Fonction addition modulo MOD de polynomes sur Device (GPU) avec multiples opérations par thread
__global__ void add_mod_multhd(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn offset, sfixn op_thread){
  sfixn i,r;
  sfixn ind=op_thread*threadIdx.x+blockIdx.x*blockDim.x*op_thread;  //blockDim.x correspond au nombre de threads par block
  for(i=0;i<op_thread;i++){
    if(ind+offset+i < size){
      r = a[ind+offset+i]+b[ind+offset+i];
      MOD_PERCENT(p,r);
      res[ind+offset+i]=r;
    }
  } 
}

__global__ void mult_tat_mod_multhd(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn size, sfixn offset, sfixn op_thread){
  sfixn i;
  sfixn ind=op_thread*threadIdx.x+blockIdx.x*blockDim.x*op_thread;  //blockDim.x correspond au nombre de threads par block
  for(i=0;i<op_thread;i++){
    if(ind+offset+i < size){
      res[ind+offset+i]=(a[ind+offset+i]*b[ind+offset+i])%p;
      if(ind+offset+i == size-1){
        printf("res=%d\n",res[ind+offset+i]);
      }
    }
  } 
}
