#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"
#include "stdint.h"


// Borne superieure pour les coefficients des polynomes
#define MAX_COEF (2048*2048)

// Threads par bloc
#define THREADS_PER_BLOCK 1024

/* Nombre de CUDA Cores sur GPU:
GPU1: nVidia GeForce GTX TITAN 2688 Cuda cores
GPU2: nVidia Tesla K40c 2880 Cuda cores
GPU3: nVidia Tesla P100-PCIe 3584 Cuda cores */
#define CUDA_CORES 2688

// Nombre de blocs necessaires pour l'appel de fonction CUDA, le nombre de blocs dépend du degré


// Degré des polynômes
//#define DEG 1000000000

// Caractéristique du corps fini auquel appartiennent les coefficients
#define MOD 65521

// Indique si l'addition doit se faire modulo MOD
#define IFMOD 1
#define D 3072

// Nombre d'opérations par thread




// Permet de dynamiser le type utilisé pour les opérations
typedef int sfixn;

// uint64_t bug à partir de 1<<28 environ
// unsigned int bug à partir de 1<<29 environ
//long bug a 500000000
sfixn OPETHD=2;
sfixn DEG=500000000;
sfixn NB_BLOCK= ((DEG+THREADS_PER_BLOCK*OPETHD-1)/(THREADS_PER_BLOCK*OPETHD));
//5254002 525500000 


// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset, sfixn op_thread){
	sfixn i;
	sfixn ind=op_thread*threadIdx.x+blockIdx.x*blockDim.x*op_thread;  //blockDim.x correspond au nombre de threads par block 
	for(i=0;i<op_thread;i++){
		if(ind+offset+i < deg){
			res[ind+offset+i]=(a[ind+offset+i]+b[ind+offset+i])%p;
			if(ind+offset+i == deg-1){
				printf("res=%d\n",res[ind+offset+i]);
				}
		}
	}
	
}

// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset, sfixn op_thread){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	
	if(ind+offset < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
	}
	
}

// Fonction d'affichage de polynome
void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i==9)
			printf("P[%d]=%d  ",i,res[i]);
		if(i==DEG-1)
			printf("P[%d]=%d  ",i,res[i]);
		if (IFMOD && res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
		}
	}
	printf("\n");
}


// main
int main(){
	sfixn *a, *b,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i, cut,offset;
	clock_t temps;
	sfixn size=DEG*sizeof(sfixn);
	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
  offset=0;
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	sfixn test=(a[DEG-1]+b[DEG-1]);
	sfixn test2=(a[0]+b[0])%MOD;
	sfixn test3=(a[D]+b[D])%MOD;
	if(IFMOD) test = test%MOD;
	printf("res[%d]=%d\n",0,test2);
	printf("res[%d]=%d\n",D,test3);
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);


	/*Appel de fonction sur le GPU */
	/* 
	  DEG+THREAD_PER_BLOCK-1 : -permet de ne pas rajouter un block si CUDA_CORES est un multiple de THREAD_PER_BLOCK
					  -permet de rajouter un block si DEG n'est pas un multiple de THREAD_PER_BLOCK	

	*/

	if(NB_BLOCK>CUDA_CORES){
      cut=NB_BLOCK/CUDA_CORES;
      if((NB_BLOCK%CUDA_CORES) != 0){
	      cut++;
	    }
      printf("Le nombre de blocs necessaire est de %d.\nIl est superieur au nombre de coeur du GPU qui est de %d.\nOn doit donc diviser les polynomes en %d parties afin de ne pas depasser le nombre de coeurs maximal.\n",NB_BLOCK,CUDA_CORES,cut);
	    printf("cut=%d\n",cut);
	    if(IFMOD){
	        for(i=0;i<cut;i++){
             offset=i*CUDA_CORES*THREADS_PER_BLOCK*OPETHD;
             if(i==cut-1){
							 printf("offset = %d\n",offset);              
							 int nb_block_restant=(((DEG-offset)+(THREADS_PER_BLOCK*OPETHD)-1)/(THREADS_PER_BLOCK*OPETHD));
							 printf("nb_block_restant = %d\n",nb_block_restant); 
							 add_mod<<<nb_block_restant,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset,OPETHD);
						 }else{
		         	 add_mod<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset,OPETHD);
         		 }
         }   
	    }else{
	    		printf("IFMOD =0\n");
	        for(i=0;i<cut;i++){
		        offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		        add<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,offset,OPETHD);
	         }  
        }
	}else{
  	printf("Le nombre de blocs necessaire est de %d.\nIl est inferieur au nombre de coeur du GPU qui est de %d.\nOn peut donc sommer les coefficients du polynome en 1 seule fois.\n",NB_BLOCK,CUDA_CORES);
		if(IFMOD==1){
    	add_mod<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,0,OPETHD);
    }else{
      add<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,0,OPETHD);
    	}	    
		}
 
	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace alloué sur le GPU */
  //affichage_polynome(res);
  printf("P[0]=%d\n",res[0]);
  printf("P[%d]=%d\n",D,res[D]);
	printf("P[%d]=%d\n",DEG-1,res[DEG-1]);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
