#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"


#define DEG 32767//ne marche plus a 100 000 000
#define MAX_COEF (2048*2048)
#define THREADS_PER_BLOCK 1025

typedef int sfixn;

__global__ void add(sfixn* a, sfixn* b, sfixn *p, sfixn* res, sfixn deg){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	if(ind < deg){
		res[ind]=(a[ind]+b[ind])%(*p);
	}
	
}

void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10)
			printf("P[%d]=%d  ",i,res[i]);
		if (res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  ",i,res[i]);
		}
	}
	printf("\n");
}

int main(){
	sfixn *a, *b, p,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i;	
	clock_t temps;
	sfixn size=DEG*sizeof(sfixn);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	sfixn test=(a[DEG-1]+b[DEG-1])%65521;
	printf("test=%d\n",test);
	/*Le modulo p un nombre premier*/	
	p=65521;

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(g_p, &p, sizeof(sfixn), hipMemcpyHostToDevice);	


	/*Appel de fonction sur le GPU */
	add<<<(DEG+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_p,g_res,DEG);

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace aloué sur le GPU */
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
