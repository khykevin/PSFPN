#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"


<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
<<<<<<< HEAD
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
// Borne superieure pour les coefficients des polynomes
#define MAX_COEF (2048*2048)

// Threads par bloc
#define THREADS_PER_BLOCK 1024

/* Nombre de CUDA Cores sur GPU:
GPU1: nVidia GeForce GTX TITAN 2688 Cuda cores
GPU2: nVidia Tesla K40c 2880 Cuda cores
GPU3: nVidia Tesla P100-PCIe 3584 Cuda cores */
#define CUDA_CORES 2688

// Nombre de blocs pour l'appel de fonction CUDA
#define NB_BLOCK ((CUDA_CORES+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)//ne marche plus a 100 000 000

// Degré des polynômes
#define DEG 6000000

// Caractéristique du corps fini auquel appartiennent les coefficients
#define MOD 65521

// Indique si l'addition doit se faire modulo MOD
#define IFMOD 1

// Nombre d'opérations par thread
#define OPETHD 1


// Permet de dynamiser le type utilisé pour les opérations
typedef int sfixn;


// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if((ind+offset)==2000001)
		printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if(ind+offset < deg){
		res[ind+offset]=(a[ind+offset]+b[ind+offset])%p;
		
	}
	
}

// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if(ind < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
=======
#define DEG 32767//ne marche plus a 100 000 000
#define MAX_COEF (2048*2048)
#define THREADS_PER_BLOCK 1025

typedef int sfixn;

__global__ void add(sfixn* a, sfixn* b, sfixn *p, sfixn* res, sfixn deg){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	if(ind < deg){
		res[ind]=(a[ind]+b[ind])%(*p);
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
	}
	
}

<<<<<<< HEAD
// Fonction d'affichage de polynome
=======
<<<<<<< HEAD
// Fonction d'affichage de polynome
=======
<<<<<<< HEAD
// Fonction d'affichage de polynome
=======
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10)
			printf("P[%d]=%d  ",i,res[i]);
		if (res[i]>65521){
			printf("ERREUR!!!\n");
<<<<<<< HEAD
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
=======
<<<<<<< HEAD
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
=======
<<<<<<< HEAD
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
=======
			printf("P[%d]=%d  ",i,res[i]);
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
		}
	}
	printf("\n");
}

<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
<<<<<<< HEAD
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe

// main
int main(){
	sfixn *a, *b,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i, cut,offset;
<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
=======
int main(){
	sfixn *a, *b, p,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i;	
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
	clock_t temps;
	sfixn size=DEG*sizeof(sfixn);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
<<<<<<< HEAD
    offset=0;
=======
<<<<<<< HEAD
    offset=0;
=======
<<<<<<< HEAD
    offset=0;
=======
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
<<<<<<< HEAD
	sfixn test=(a[DEG-1]+b[DEG-1])%MOD;
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	
=======
<<<<<<< HEAD
	sfixn test=(a[DEG-1]+b[DEG-1])%MOD;
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	
=======
<<<<<<< HEAD
	sfixn test=(a[DEG-1]+b[DEG-1])%MOD;
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	
=======
	sfixn test=(a[DEG-1]+b[DEG-1])%65521;
	printf("test=%d\n",test);
	/*Le modulo p un nombre premier*/	
	p=65521;
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
<<<<<<< HEAD
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe


	/*Appel de fonction sur le GPU */
	/* 
	  CUDA_CORES+THREAD_PER_BLOCK-1 : -permet de ne pas rajouter un block si CUDA_CORES est un multiple de THREAD_PER_BLOCK
					  -permet de rajouter un block si CUDA_CORES n'est pas un multiple de THREAD_PER_BLOCK	

	*/
	if(DEG > CUDA_CORES*THREADS_PER_BLOCK){
	  cut = DEG/(CUDA_CORES*THREADS_PER_BLOCK);
	  if(DEG % (CUDA_CORES*THREADS_PER_BLOCK) != 0){
	    cut++;
	  }
	} else {
	  cut = 1;
	}
	printf("cut=%d\n",cut);
	if(IFMOD){
	  for(i=0;i<cut;i++){
	    offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		printf("cut = %d, offset = %d\n",cut,offset);    
		add_mod<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset);
	  }   
	}else{
	  for(i=0;i<cut;i++){
		offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		add<<<CUDA_CORES*THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,offset);
	  }  
    }
<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
=======
	hipMemcpy(g_p, &p, sizeof(sfixn), hipMemcpyHostToDevice);	


	/*Appel de fonction sur le GPU */
	add<<<(DEG+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_p,g_res,DEG);
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

<<<<<<< HEAD
	/*Liberation de l'espace alloué sur le GPU */
=======
<<<<<<< HEAD
	/*Liberation de l'espace alloué sur le GPU */
=======
<<<<<<< HEAD
	/*Liberation de l'espace alloué sur le GPU */
=======
	/*Liberation de l'espace aloué sur le GPU */
>>>>>>> 8d1f4218625334a38fa83452e47fa45b12a10edc
>>>>>>> d46cf46a4e0d87f915af7d98586f1b6ba5af52d9
>>>>>>> e60feb9dcfbc12986b018024d33f0a87ed1946fe
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
