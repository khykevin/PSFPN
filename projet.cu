#include "hip/hip_runtime.h"
#include "sys/time.h"
#include "time.h"
#include "global_fun.h"

// Threads par bloc
#define THREADS_PER_BLOCK 1024

/* Nombre de CUDA Cores sur GPU:
GPU1: nVidia GeForce GTX TITAN 2688 Cuda cores
GPU2: nVidia Tesla K40c 2880 Cuda cores
GPU3: nVidia Tesla P100-PCIe 3584 Cuda cores */
#define CUDA_CORES 2688

// Nombre de blocs necessaires pour l'appel de fonction CUDA, le nombre de blocs dépend du degré
#define NB_BLOCK ((DEG+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)//ne marche plus a 100 000 000

// Degré des polynômes
//#define DEG 1000000000

// Caractéristique du corps fini auquel appartiennent les coefficients
#define MOD 65521

// Indique si l'addition doit se faire modulo MOD
#define IFMOD 1

// Nombre d'opérations par thread
#define OPETHD 1


// uint64_t bug à partir de 1<<28 environ
// unsigned int bug à partir de 1<<29 environ
//long bug a 500000000

sfixn DEG=500000000;
//5254002 525500000 

// Fonction d'affichage de polynome
void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i==DEG-1)
			printf("P[%d]=%d  ",i,res[i]);
		if (IFMOD && res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
		}
	}
	printf("\n");
}

// main
int main(){
	sfixn *a, *b,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i, cut,offset;
	clock_t temps;
	sfixn size=DEG*sizeof(sfixn);
	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
  offset=0;
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) ((MOD-1)*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) ((MOD-1)*((double)rand())/ RAND_MAX);
	}
	sfixn test=(a[DEG-1]+b[DEG-1]);
	sfixn test2=(a[0]+b[0])%MOD;
	if(IFMOD) test = test%MOD;
	printf("res[%d]=%d\n",0,test2);
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);


	/*Appel de fonction sur le GPU */
	/* 
	  CUDA_CORES+THREAD_PER_BLOCK-1 : -permet de ne pas rajouter un block si CUDA_CORES est un multiple de THREAD_PER_BLOCK
					  -permet de rajouter un block si CUDA_CORES n'est pas un multiple de THREAD_PER_BLOCK	

	*/

	if(NB_BLOCK>CUDA_CORES){
      cut=NB_BLOCK/CUDA_CORES;
      if((NB_BLOCK%CUDA_CORES) != 0){
	      cut++;
	    }
      printf("Le nombre de blocs necessaire est de %d.\nIl est superieur au nombre de coeur du GPU qui est de %d.\nOn doit donc diviser les polynomes en %d parties afin de ne pas depasser le nombre de coeurs maximal.\n",NB_BLOCK,CUDA_CORES,cut);
	    printf("cut=%d\n",cut);
	    if(IFMOD){
	        for(i=0;i<cut;i++){
             offset=i*CUDA_CORES*THREADS_PER_BLOCK;
             if(i==cut-1){
							 int nb_block_restant=(((DEG-offset)+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK);
							 add_mod<<<nb_block_restant,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset);
						 }else{
		         	 add_mod<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset);
         		 }
         }   
	    }else{
	    		printf("IFMOD =0\n");
	        for(i=0;i<cut;i++){
		        offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		        add<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,offset);
	         }  
        }
	}else{
  	printf("Le nombre de blocs necessaire est de %d.\nIl est inferieur au nombre de coeur du GPU qui est de %d.\nOn peut donc sommer les coefficients du polynome en 1 seule fois.\n",NB_BLOCK,CUDA_CORES);
		if(IFMOD==1){
    	add_mod<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,0);
    	}else{
      	add<<<NB_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,0);
    	}	    
		}
 
	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace alloué sur le GPU */
  //affichage_polynome(res);
  printf("P[0]=%ld\n",res[0]);
	printf("P[%d]=%ld\n",DEG-1,res[DEG-1]);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
