#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"
/* Traitement pour le modulo p
Probleme avec le changement du nombre de threads cela engendre une resultat errone

*/

#define DEG 1024
#define MAX_COEF 2048*2048

__global__ void add(int* a, int* b, int *p, int* res){
	res[threadIdx.x]=(a[threadIdx.x]+b[threadIdx.x])%(*p);
}

void affichage_polynome(int *res){
	int i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10){
			printf("P[%d]=%d  ",i,res[i]);
		}
		if (res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  ",i,res[i]);
		}
	}
	printf("\n");
}


int main(){
	int *a, *b, p,*res; /*Copie des variables sur CPU, p*/
	int *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	int i;	
	clock_t begin = clock();


	int size=DEG*sizeof(int);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(int));
	hipMalloc((void**)&g_res, size);
	
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	res = (int*)malloc(size);
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(int) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(int) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	/*Le modulo p un nombre premier*/	
	p=65521;

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(g_p, &p, sizeof(int), hipMemcpyHostToDevice);	


	/*Appel de fonction sur le GPU */
	add<<<1,DEG>>>(g_a,g_b,g_p,g_res);

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace aloué sur le GPU */
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Le temps d'execution est de : %g\n",time_spent);
	return 0;
}
