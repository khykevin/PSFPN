#include "hip/hip_runtime.h"
#include "global_fun.h"
#include "types.h"


// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block 
	if(ind+offset < deg){
		res[ind+offset]=(a[ind+offset]+b[ind+offset])%p; 
	}
	
}

// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	
	if(ind+offset < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
	}
	
}


