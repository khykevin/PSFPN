#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"


#define DEG 32767//ne marche plus a 100 000 000
#define MAX_COEF (2048*2048)
#define THREADS_PER_BLOCK 1024

__global__ void add(int* a, int* b, int *p, int* res, int deg){
	int ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	if(ind < deg){
		res[ind]=(a[ind]+b[ind])%(*p);
	}
	
}

void affichage_polynome(int *res){
	int i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10)
			printf("P[%d]=%d  ",i,res[i]);
		if (res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  ",i,res[i]);
		}
	}
	printf("\n");
}

int main(){
	int *a, *b, p,*res; /*Copie des variables sur CPU, p*/
	int *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	int i;	
	clock_t temps;
	int size=DEG*sizeof(int);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(int));
	hipMalloc((void**)&g_res, size);
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	res = (int*)malloc(size);
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(int) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(int) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	int test=(a[DEG-1]+b[DEG-1])%65521;
	printf("test=%d\n",test);
	/*Le modulo p un nombre premier*/	
	p=65521;

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(g_p, &p, sizeof(int), hipMemcpyHostToDevice);	


	/*Appel de fonction sur le GPU */
	add<<<(DEG+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_p,g_res,DEG);

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace aloué sur le GPU */
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
