#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"


// Borne superieure pour les coefficients des polynomes
#define MAX_COEF (2048*2048)

// Threads par bloc
#define THREADS_PER_BLOCK 1024

/* Nombre de CUDA Cores sur GPU:
GPU1: nVidia GeForce GTX TITAN 2688 Cuda cores
GPU2: nVidia Tesla K40c 2880 Cuda cores
GPU3: nVidia Tesla P100-PCIe 3584 Cuda cores */
#define CUDA_CORES 2688

// Nombre de blocs pour l'appel de fonction CUDA
#define NB_BLOCK ((CUDA_CORES+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)//ne marche plus a 100 000 000

// Degré des polynômes
#define DEG 6000000

// Caractéristique du corps fini auquel appartiennent les coefficients
#define MOD 65521

// Indique si l'addition doit se faire modulo MOD
#define IFMOD 1

// Nombre d'opérations par thread
#define OPETHD 1


// Permet de dynamiser le type utilisé pour les opérations
typedef int sfixn;


// Fonction addition modulo MOD de polynomes sur Device (GPU)
__global__ void add_mod(sfixn* a, sfixn* b, sfixn p, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if((ind+offset)==2000001)
		printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if(ind+offset < deg){
		res[ind+offset]=(a[ind+offset]+b[ind+offset])%p;
		
	}
	
}

// Fonction addition de polynomes sans modulo sur Device (GPU)
__global__ void add(sfixn* a, sfixn* b, sfixn* res, sfixn deg, sfixn offset){
	sfixn ind=threadIdx.x+blockIdx.x*blockDim.x;  //blockDim.x correspond au nombre de threads par block
	//printf("threadIdx.x = %d, blockIdx = %d, blockDim.x = %d, indice=%d\n",threadIdx.x,blockIdx.x,blockDim.x, ind);
	if(ind < deg){
		res[ind+offset]=a[ind+offset]+b[ind+offset];
	}
	
}

// Fonction d'affichage de polynome
void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10)
			printf("P[%d]=%d  ",i,res[i]);
		if (res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  \n",i,res[i]);
			exit(0);
		}
	}
	printf("\n");
}


// main
int main(){
	sfixn *a, *b,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i, cut,offset;
	clock_t temps;
	sfixn size=DEG*sizeof(sfixn);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
    offset=0;
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	sfixn test=(a[DEG-1]+b[DEG-1])%MOD;
	printf("res[%d]=%d\n",DEG-1,test);
	/*Le modulo p un nombre premier*/	

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);


	/*Appel de fonction sur le GPU */
	/* 
	  CUDA_CORES+THREAD_PER_BLOCK-1 : -permet de ne pas rajouter un block si CUDA_CORES est un multiple de THREAD_PER_BLOCK
					  -permet de rajouter un block si CUDA_CORES n'est pas un multiple de THREAD_PER_BLOCK	

	*/
	if(DEG > CUDA_CORES*THREADS_PER_BLOCK){
	  cut = DEG/(CUDA_CORES*THREADS_PER_BLOCK);
	  if(DEG % (CUDA_CORES*THREADS_PER_BLOCK) != 0){
	    cut++;
	  }
	} else {
	  cut = 1;
	}
	printf("cut=%d\n",cut);
	if(IFMOD){
	  for(i=0;i<cut;i++){
	    offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		printf("cut = %d, offset = %d\n",cut,offset);    
		add_mod<<<CUDA_CORES,THREADS_PER_BLOCK>>>(g_a,g_b,MOD,g_res,DEG,offset);
	  }   
	}else{
	  for(i=0;i<cut;i++){
		offset=i*CUDA_CORES*THREADS_PER_BLOCK;
		add<<<CUDA_CORES*THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(g_a,g_b,g_res,DEG,offset);
	  }  
    }

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace alloué sur le GPU */
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	temps=clock();
	printf("Le temps d'execution est de : %f\n",(double)temps/CLOCKS_PER_SEC);
	return 0;
}
