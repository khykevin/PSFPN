#include "stdio.h"
#include "stdlib.h"
#include "sys/time.h"
#include "time.h"
#include "hip/hip_runtime.h"
/* Traitement pour le modulo p
Probleme avec le changement du nombre de threads cela engendre une resultat errone

*/

#define DEG 32767
#define MAX_COEF 2048*2048

typedef int sfixn;

__global__ void add(sfixn* a, sfixn* b, sfixn *p, sfixn* res){
	res[blockIdx.x]=(a[blockIdx.x]+b[blockIdx.x])%(*p);
}

void affichage_polynome(sfixn *res){
	sfixn i;
	for(i=0;i<DEG;i++){
		if(i>DEG-10){
			printf("P[%d]=%d  ",i,res[i]);
		}
		if (res[i]>65521){
			printf("ERREUR!!!\n");
			printf("P[%d]=%d  ",i,res[i]);
		}
	}
	printf("\n");
}


int main(){
	sfixn *a, *b, p,*res; /*Copie des variables sur CPU, p*/
	sfixn *g_a, *g_b,*g_p,*g_res; /*Copie des variables sur GPU, g_p*/
	sfixn i;	
	clock_t begin = clock();


	sfixn size=DEG*sizeof(sfixn);	

	/*On alloue les vecteur de coefficients sur le GPU*/	
	hipMalloc((void**)&g_a, size);
	hipMalloc((void**)&g_b, size);
	hipMalloc((void**)&g_p, sizeof(sfixn));
	hipMalloc((void**)&g_res, size);
	
	a = (sfixn*)malloc(size);
	b = (sfixn*)malloc(size);
	res = (sfixn*)malloc(size);
	srand(time(NULL));
	/* On initialise les coefficients de polynomes de degre DEG */
	for(i=0; i<DEG; i++){
		a[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
		b[i]=(sfixn) (MAX_COEF*((double)rand())/ RAND_MAX);
	}
	/*Le modulo p un nombre premier*/	
	p=65521;

	/*On copie sur le GPU les vecteurs initialisé sur le CPU*/
	hipMemcpy(g_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(g_p, &p, sizeof(sfixn), hipMemcpyHostToDevice);	


	/*Appel de fonction sur le GPU */
	add<<<DEG,1>>>(g_a,g_b,g_p,g_res);

	/*Copie du resultat du GPU sur le CPU*/
	hipMemcpy(res, g_res, size, hipMemcpyDeviceToHost);

	/*Liberation de l'espace aloué sur le GPU */
	affichage_polynome(res);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_p);
	hipFree(g_res);
	
	free(a);
	free(b);
	free(res);
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Le temps d'execution est de : %g\n",time_spent);
	return 0;
}
